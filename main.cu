#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

template<int kThreadCount>
__global__ void Kernel(const int* values, int count, int* global_min) {

  // CUDA programming guide 8.4.2.1: Thread Block Tile
  //
  // From the notes, thread blocks larger than 32 need a small amount
  // of shared memory allocated for CC <= 7.5.
  __shared__ cg::block_tile_memory<kThreadCount> for_reduction;
  cg::thread_block thread_block = cg::this_thread_block(for_reduction);

  // Calculate thread-local minimum value.
  int minimum = std::numeric_limits<int>::max();
  for(int x = thread_block.thread_rank(); x < count; x += kThreadCount) {
    int v = values[x];
    minimum = min(v, minimum);
  }

  // Calculate the minimum across all threads in the block.
  minimum = cg::reduce(cg::tiled_partition<kThreadCount>(thread_block), minimum, cg::less<int>());
  if(thread_block.thread_rank() == 0) {
    global_min[0] = minimum;
  }
}

// Thank you StackOverflow.
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CHECK_CUDA(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int, char**) {

  // Generate some random integer values in managed memory.
  static const int kSize = 5000;
  int* samples;
  CHECK_CUDA(hipMallocManaged(&samples, sizeof(int) * kSize));
  for(int i = 0; i < kSize; i++) samples[i] = (rand() % 100) + 10;

  // Allocate output.
  int* gpu_min;
  CHECK_CUDA(hipMallocManaged(&gpu_min, sizeof(int)));

  // Calculate the minimum.
  static const int kThreadCount = 256;
  Kernel<kThreadCount><<<1, kThreadCount>>>(samples, kSize, gpu_min);
  CHECK_CUDA(hipDeviceSynchronize());

  // Check that the min matches the host minimum.
  int host_min = *std::min_element(samples, samples + kSize);
  std::cout << "Host min = " << host_min << "; Device min = " << *gpu_min << std::endl;
  if(host_min == *gpu_min) {
    std::cout << "Host and device calculation match - we're good!" << std::endl;
    return EXIT_SUCCESS;
  } else {
    std::cout << "Host and device calculation don't match - there's a problem here." << std::endl;
    return EXIT_FAILURE;
  }
}